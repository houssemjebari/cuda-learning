#include <hip/hip_runtime.h>
#include <stdio.h>

// Our mutex structure
struct Mutex {
    int * lock;
};

// Initialize the mutex 
__host__ void initMutex(Mutex * m) {
    hipMalloc((void **) &m->lock, sizeof(int));
    int initial = 0;
    hipMemcpy(m->lock, &initial, sizeof(int), hipMemcpyHostToDevice);
}

// Acquire the mutex 
__device__ void lock(Mutex *m) {
    while (atomicCAS(m->lock, 0 , 1) != 0) {
    }
}

// Release the mutex 
__device__ void unlock(Mutex *m) {
    atomicExch(m->lock, 0);
}

// Kernel function to demonstrate the mutex usage
__global__ void mutexKernel(int* counter, Mutex *m) {
    lock(m);
    // Critical section
    int old = *counter;
    *counter = old + 1;
    unlock(m);
}

int main() {
    Mutex m;
    initMutex(&m);

    int *d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    int initial = 0;
    hipMemcpy(d_counter, &initial, sizeof(int), hipMemcpyHostToDevice);

    // Launch Kernel with multiple threads 
    mutexKernel<<<1, 1000>>>(d_counter, &m);

    // Copy the results back from the GPU
    int result;
    hipMemcpy(&result, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("Counter value: %d\n", result);

    // Free-up memory
    hipFree(m.lock);
    hipFree(d_counter);
}